#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_cd1.cuh"
#include "../include/dervfields_cd1.cuh"
#include "../include/usersource_cd1.cuh"

__device__ __host__
int divflux1(real *dw, real *wd, real *w, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;

dw[fencode3_cd1(p,ii,field)]+= grad3dn_cd1(wd,wd,p,ii,flux,dir); 
//dw[fencode3_cd1(p,ii,field)]+= grad3d_cd1(wd,p,ii,flux,dir); 
//dw[fencode3_cd1(p,ii,field)]=0.0;
 /*switch(field)
  {

     case mom1:
       dw[fencode3_cd1(p,ii,field)]-= (p->g[dir])*w[fencode3_cd1(p,ii,rho)];
      break;
    case mom2:
      dw[fencode3_cd1(p,ii,field)]-= (p->g[dir])*w[fencode3_cd1(p,ii,rho)];
      break;
#if defined USE_SAC_3D
    case mom3:
      dw[fencode3_cd1(p,ii,field)]-= (p->g[dir])*w[fencode3_cd1(p,ii,rho)];
      break;
#endif
    case rho:
     ;// dw[fencode3_cd1(p,ii,field)]+= ix/800;
      break;

  } */   
 // dw[fencode3_cd1(p,ii,field)]= gradd0_cd1(wd,p,ii,f1,0)+gradd1_cd1(wd,p,ii,f2,1);    
  return ( status);
}






__device__ __host__
real transportflux (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  

   //real fluxt=0;

   //transport flux
   //this will work without the switch as follows
        #if defined USE_SAC || defined USE_SAC_3D
     return(w[fencode3_cd1(p,ii,mom1+direction)]*w[fencode3_cd1(p,ii,field)]/(w[fencode3_cd1(p,ii,rho)]+w[fencode3_cd1(p,ii,rhob)]));

        #else
     return(w[fencode3_cd1(p,ii,mom1+direction)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)]);

        #endif


  
 
}






__device__ __host__
real fluxmom1 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


         #if defined USE_SAC || defined USE_SAC_3D
     		return( -(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}




__device__ __host__
real fluxmom10 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


         #if defined USE_SAC || defined USE_SAC_3D
         return(direction==0?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}

__device__ __host__
real fluxmom11 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


         #if defined USE_SAC || defined USE_SAC_3D
         return(direction==1?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}


__device__ __host__
real fluxmom12 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


         #if defined USE_SAC || defined USE_SAC_3D
         return(direction==2?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}







__device__ __host__
int computefluxrho (real *dw, real *wd, real *w, struct params *p,int *ii,int direction) {

  int field;
  int status=0;
      wd[fencode3_cd1(p,ii,flux)]=0.0;
  // for(direction=0;direction<2;direction++)
         #if defined USE_SAC || defined USE_SAC_3D
	      wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,rho,direction)+(w[fencode3_cd1(p,ii,rhob)]*w[fencode3_cd1(p,ii,mom1+direction)])/(w[fencode3_cd1(p,ii,rhob)]+w[fencode3_cd1(p,ii,rho)]);
         #else
             wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,rho,direction);
         #endif
  
  return ( status);
}


__device__ __host__
int computefluxmom3 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

#ifdef USE_SAC_3D
               wd[fencode3_cd1(p,ii,flux)]=0.0;


    		wd[fencode3_cd1(p,ii,flux)]+=transportflux(dw,wd,w,p,ii,field,direction)+fluxmom12(dw,wd,w,p,ii,field,direction);
               

#endif

  return ( status);
}




__device__ __host__
int computefluxmom2 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

               wd[fencode3_cd1(p,ii,flux)]=0.0;
 
        #ifdef USE_SAC
    		wd[fencode3_cd1(p,ii,flux)]+= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom11(dw,wd,w,p,ii,field,direction);

 
        #endif
        #ifdef USE_SAC_3D
    		wd[fencode3_cd1(p,ii,flux)]+= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom11(dw,wd,w,p,ii,field,direction);
 
 
        #endif

  return ( status);
}







__device__ __host__
int computefluxmom1 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

               wd[fencode3_cd1(p,ii,flux)]=0.0;



        #ifdef ADIABHYDRO
     		wd[fencode3_cd1(p,ii,flux)]+= transportflux(dw,wd,w,p,ii,field,direction);
        #endif
        #ifdef USE_SAC
    		wd[fencode3_cd1(p,ii,flux)]+= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom10(dw,wd,w,p,ii,field,direction);
 
        #endif
        #ifdef USE_SAC_3D
    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom10(dw,wd,w,p,ii,field,direction);
 
        #endif
        
  return ( status);
}







//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case rho:
      //computevel3_cd1(w,wd,p,ii);
      //computept3_cd1(w,wd,p,ii);
      computefluxrho(dw,wd,w,p,ii,dir);
     break;
     case mom1:
      computefluxmom1(dw,wd,w,p,ii,field,dir);
      //wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,pressuret)];
     break;
     case mom2:
       computefluxmom2(dw,wd,w,p,ii,field,dir);
       //wd[fencode3_cd1(p,ii,f2)]+=wd[fencode3_cd1(p,ii,pressuret)];
     break;
     #ifdef USE_SAC_3D
       case mom3:
        computefluxmom3(dw,wd,w,p,ii,field,dir);
        //wd[fencode3_cd1(p,ii,f3)]+=wd[fencode3_cd1(p,ii,pressuret)];
       break;
     #endif
  }
  //return ( status);
}




__global__ void centdiff1init_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     
 

   fid=0;
   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd1(p,ii,f)]=0.0;
                  	    //for(fid=0;fid<2;fid++)
                               wd[fencode3_cd1(p,ii,flux)]=0.0;
                               //wmod[fencode_cd1(p,i,j,flux)+order*NVAR*(p->n[0])*(p->n[1])]=0.0;
                               /*wmod[fencode3_cd1(p,ii,b1b)+(ordero*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,b1b)+(order*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,b2b)+(ordero*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,b2b)+(order*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,b3b)+(ordero*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,b3b)+(order*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,energyb)+(ordero*NVAR*dimp)]=0.0;
                               wmod[fencode3_cd1(p,ii,energyb)+(order*NVAR*dimp)]=0.0;*/
                        }

   
 __syncthreads();                       




}



__global__ void centdiff1_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


//1. 11/1/11 could swap cases below
                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,0); 
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,1); 
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,2); 
                         break;
                         #endif
                        }
              //  }
                        //might need to set boundaries correctly
 

__syncthreads();                        



}










__global__ void centdiff1a_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   fid=0;





     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

			// if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
			     #ifdef USE_SAC
				   if(ii[0]>1 && ii[1] >1 && ii[0]<(ni-2) && ii[1]<(nj-2))
			     #endif
			     #ifdef USE_SAC_3D
				  if(ii[0]>1 && ii[1] >1 && ii[2] >1 && ii[0]<(ni-2) && ii[1]<(nj-2) && ii[2]<(nk-2))
			     #endif                        
                               divflux1(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir);  


 __syncthreads();


}

__global__ void centdiff1af_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

             // for(int f=rho; f<=mom3; f++)
              // {



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


                        switch(dir)
                        {
                         case 0:

                         //if(i<(ni)  && j >1 &&  j<(nj-2))
			     #ifdef USE_SAC
				   if(ii[1]>1 && ii[1] <(nj-2) && ii[0]<(ni) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[1]>1 && ii[1] <(nj-2) && ii[0]<(ni) &&  ii[2]>1 && ii[2] <(nk-2) )
			     #endif                          
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)]; 
                         break;
                         case 1:
			     #ifdef USE_SAC
				   if(ii[0]>1 && ii[1] <(nj) && ii[0]<(ni-2) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[0]>1 && ii[1] <(nj) && ii[0]<(ni-2) &&  ii[2]>1 && ii[2] <(nk-2) )
			     #endif 
                         
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];
                         break;
                         case 2:

 
			     #ifdef USE_SAC_3D
				   if(ii[0]>1 &&  ii[0]<(ni-2)  && ii[1]>1 &&  ii[1]<(nj-2) && ii[2] <(nk) )
                               wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];
			     #endif                         
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                             
                         break;
                        }


              //  }
	

  __syncthreads();


}


__global__ void centdiff1binit_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     
 

   fid=0;
   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd1(p,ii,f)]=0.0;

                        }

   
 __syncthreads();                       




}


__global__ void centdiff1b_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   fid=0;





     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


#if(defined(USE_USERSOURCE))
   {

     ii[0]=ip;
     ii[1]=jp;
#endif
     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
	   ii[2]=kp;
     #endif


     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
       //if(ii[0]<((p->n[0])-2) && ii[1]<((p->n[1])-2) && ii[2]<((p->n[2])-2)     && ii[0]>1    &&  ii[1]>1   && ii[2]>1   )
       if(ii[0]<((p->n[0])) && ii[1]<((p->n[1])) && ii[2]<((p->n[2]))    )
     #endif
     #if(defined(USE_SAC) && defined(USE_USERSOURCE))
       //if(ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)
      if(ii[0]<(p->n[0]) && ii[1]<(p->n[1]))
     #endif

                     #ifdef USE_USERSOURCE
                            addsourceterms1_cd1(dwn1,wd,wmod+ordero*NVAR*dimp,p,s,ii,f,dir); 


                      }
                    __syncthreads();
                     #endif



               // }
    


}



__global__ void centdiff1bf_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

             // for(int f=rho; f<=mom3; f++)
              // {



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


			     #ifdef USE_SAC
				   if( ii[1] <(nj) && ii[0]<(ni) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[1] <(nj) && ii[0]<(ni) &&   ii[2] <(nk) )
			     #endif                          
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];

            

  __syncthreads();


}



/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int cucentdiff1(struct params **p, struct params **d_p,struct state **d_s, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt, int field, int dir)
{
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    //printf("gamma %g\n", (*p)->gamma);
     centdiff1init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     centdiff1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     centdiff1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     hipDeviceSynchronize();


     centdiff1af_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     hipDeviceSynchronize();
     
    // centdiff1binit_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
    // hipDeviceSynchronize();
     
   //  centdiff1b_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
   //  hipDeviceSynchronize();
     
   //  centdiff1bf_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
   //  hipDeviceSynchronize();
     
}


