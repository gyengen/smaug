#include "hip/hip_runtime.h"
//#define MODID pre


#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/step.h"
#include "../include/gradops_cdf.cuh"
#include "../include/dervfields_cdf.cuh"
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void computevels_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     









  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         #endif
                        }


         }


              __syncthreads();











  
}


__global__ void computept_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                                           
                           computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           
                            computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                          
                                computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         #endif
                        }


         }




  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



  /*   ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #else
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #endif */        
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


        /* }*/


              __syncthreads();











  
}


__global__ void computeptzero_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        wd[fencode3_cdf(p,ii,pressuret)]=0.0;
                        


         }




  


              __syncthreads();











  
}



__global__ void computepk_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     









  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #else
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #endif         
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


         }


              __syncthreads();











  
}


__global__ void computepbg_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp;
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#else
   dimp=((p->n[0]))*((p->n[1]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     









  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       computepbg3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	       
	     #else
	       computepbg3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	       
	     #endif         
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


         }


              __syncthreads();











  
}


__global__ void computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     




   /*for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
        }

}*/
              __syncthreads();



if(iindex==0)
{
   
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    //p->cmax=0.0;
    for(ii[0]=0;ii[0]<((p->n[0]));ii[0]++)
      for(ii[1]=0;ii[1]<((p->n[1]));ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]=0;ii[2]<((p->n[2]));ii[2]++)
     #endif
	{ 
               computecmax3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);




	}

 //  }
}
 __syncthreads(); 

//p->cmax=1.0;



  
}

//from http://www.nvidia.com/object/cuda_sample_data-parallel.html#reduction
/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory
*/
__global__ void fastcomputemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;
        extern __shared__ real sdata[];
  // __shared__ float sdata[];
 //real sdata[dimp];
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     



    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   sdata[tid]=0.0;


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>sdata[tid])
                    sdata[tid]=wd[fencode3_cdf(p,ii,cfast)];
        }


              __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            //sdata[tid] += sdata[tid + s];
            //if(sdata[tid]>sdata[0])
             //   sdata[0]=sdata[tid];
            if(sdata[tid+s]>sdata[tid])
                sdata[tid]=sdata[tid+s];
        }
        __syncthreads();
    }


    if (tid == 0) p->cmax = sdata[0];

  /* for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))
                    sdata[tid]=wd[fencode3_cdf(p,ii,cfast)];
        }

}
              __syncthreads();*/

    // do reduction in shared mem
    /*for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            //sdata[tid] += sdata[tid + s];
            //if(sdata[tid]>sdata[0])
             //   sdata[0]=sdata[tid];
            if(sdata[tid+s]>sdata[tid])
                sdata[tid]=sdata[tid+s];
        }
        __syncthreads();
    }


    if (tid == 0) p->cmax = sdata[0];*/

  
}



/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */


//from http://www.nvidia.com/object/cuda_sample_data-parallel.html#reduction
/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */
__global__ void reduction0computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  real temp[1024];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
    if(iindex<1024)
      temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

   /*  #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif*/
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
              // if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))


         if(iindex<dimp)
                    sdata[tid]=wd[fencode3_cdf(p,ii,cfast)];

              __syncthreads();


    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {



        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
            
        }
        // strided indexing using sequential addressing is better!
        /*int tindex=2*s*tid;
        if (tindex<blockDim.x) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
        }
        __syncthreads();*/
         __syncthreads();
    }

    __syncthreads();
    if(tid==0)
      temp[blockIdx.x]=sdata[0];
__syncthreads();
    if(iindex==0)
       for(int i=0; i<numBlocks; i++)
         if(temp[i]>(p->cmax)) p->cmax=temp[i];
     if (tid == 0 && p->cmax<sdata[0] ) p->cmax = sdata[0];
 


    /* ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))
                    p->cmax=wd[fencode3_cdf(p,ii,cfast)];
        }


              __syncthreads();*/
 
//        }

//}
//p->cmax=1.0;
 
}



__global__ void myreduction0computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp,int ndimp,int s)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;


   
    //while(((s*=2)<=((ndimp/2)-1)) && ((iindex+s)<ndimp)) {
    if((iindex+s)<ndimp)
            if(temp[iindex+s]>temp[iindex])
                 temp[iindex]=temp[iindex + s];
            
       // }

       //  __syncthreads();
    

   // __syncthreads();

   if(iindex==0)
      p->cmax=temp[0];


 
}



__global__ void myreduction0computemaxcourant_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp,int ndimp,int s)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;


   
    //while(((s*=2)<=((ndimp/2)-1)) && ((iindex+s)<ndimp)) {
    if((iindex+s)<ndimp)
            if(temp[iindex+s]>temp[iindex])
                 temp[iindex]=temp[iindex + s];
            
       // }

       //  __syncthreads();
    

   // __syncthreads();

   if(iindex==0  && (p->maxcourant<temp[0]))
      p->maxcourant=temp[0];


 
}




__global__ void zeropadmaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp, int ndimp)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 
  if(iindex<ndimp)
      temp[iindex]=0.0;

}

__global__ void zeropadmaxcourant_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp, int ndimp)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 
  //if(iindex<ndimp)
  //    temp[iindex]=0.0;

  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

    
//int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
             temp[iindex]=temp[iindex]/(wd[fencode3_cdf(p,ii,delx1+dir)]);







}

__global__ void reduction0computemaxcfast_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


 // int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

    
    unsigned int iindex = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
    extern __shared__ real sdata[];



    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   sdata[tid]=0.0;

                   sdata[tid]=wd[blockDim.x+(cfast*dimp)+iindex];

              __syncthreads();


    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0 && (p->cmax)<sdata[0] ) p->cmax = sdata[0];
 __syncthreads();



 
}




//from http://www.nvidia.com/object/cuda_sample_data-parallel.html#reduction
/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */
__global__ void reductiona0computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
        extern __shared__ real sdata[];
   //__shared__ float sdata[];
 //real sdata[dimp];
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     



    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   sdata[tid]=0.0;

   if(iindex<dimp)
      sdata[tid]=wd[iindex+(dimp*cfast)];

       /* if(iindex<dimp)
               if(wd[iindex+(dimp*cfast)]>(p->cmax))
                    sdata[tid]=wd[iindex+(dimp*cfast)];*/

              __syncthreads();


    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) p->cmax = sdata[0];



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))
                    p->cmax=wd[fencode3_cdf(p,ii,cfast)];
        }


              __syncthreads();
  
}


__global__ void computec_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     






 p->cmax=0.0;


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
        }


              __syncthreads();












  
}


__global__ void computedervfields_parallel(struct params *p,   real *wmod, real *wd, int order)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     






if(order == 0)

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		

               for(int f=vel1; f<=pkb; f++)
                        wd[fencode3_cdf(p,ii,f)]=0; 
		#ifdef USE_SAC_3D
		  for(int f=rho; f<=b3; f++)
                  	wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]=wmod[fencode3_cdf(p,ii,f)]; 

		#else
		  for(int f=rho; f<=b2; f++)
                  	wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]=wmod[fencode3_cdf(p,ii,f)]; 
		#endif               

        }


               __syncthreads();



  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if( ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               
             #ifdef ADIABHYDRO
               //computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
             #else
               //computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computej3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

               computebdotv3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computedivb3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

             #endif

         }


              __syncthreads();

  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cdf(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucomputedervfields(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   ////hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computedervfields_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputevels(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   ////hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computevels_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputemaxc(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir, real **wd, real **d_wtemp)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
  
  real fn,fractn,in;
  int ndimp;
////hipSetDevice(selectedDevice);
   int nit=100;
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

    fn=log(dimp)/log(2.0);
    fractn=modf(fn,&in);
    
    if(fractn>0)
    {
       fn+=1;
       ndimp=(int)pow(2,fn);
     }
     else
       ndimp=dimp;
       


   (*p)->cmax=0.0;
    int smemSize = numThreadsPerBlock * sizeof(real);
  hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;




//hipMemcpy(*d_wtemp, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
  zeropadmaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);

hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
hipMemcpy(*d_wtemp, ((*wd)+(cfast*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
 
int s=1;
while(((s*=2)<=((ndimp/2)-1)) ) 
{
   myreduction0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp,s);
   hipDeviceSynchronize();
}


//reduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
//myreduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd,*d_wtemp, order, dir);

 //reductiona0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
  // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
    // fastcomputemaxc_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
hipDeviceSynchronize();

//(*p)->cmax=2.0;
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

//printf("cmax on device %.8f\n",(*p)->cmax);
//(*p)->cmax=0.0;
//hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
/*for(int i=0; i<dimp;i++)
{

if(((*wd)[i+(cfast*dimp)])>((*p)->cmax))
                    (*p)->cmax=(*wd)[i+(cfast*dimp)];
}
printf("cmax on cpu %.8f\n",(*p)->cmax);*/
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 /*for(int i=0; i<nit;i++)
{
 reduction0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
   // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
     hipDeviceSynchronize();
}*/



//    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

  // hipFree(*d_ttemp);
  //checkErrors("copy data from device");


 


}



int cucomputemaxcourant(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir, real **wd, real **d_wtemp)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
  
  real fn,fractn,in;
  int ndimp;
////hipSetDevice(selectedDevice);
   int nit=100;
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

    fn=log(dimp)/log(2.0);
    fractn=modf(fn,&in);
    
    if(fractn>0)
    {
       fn+=1;
       ndimp=(int)pow(2,fn);
     }
     else
       ndimp=dimp;
       


   //(*p)->maxcourant=0.0;
    int smemSize = numThreadsPerBlock * sizeof(real);
  hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;




//hipMemcpy(*d_wtemp, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
//  zeropadmaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);
hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
hipMemcpy(*d_wtemp, ((*wd)+(cfast*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
 zeropadmaxcourant_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);
int s=1;
while(((s*=2)<=((ndimp/2)-1)) ) 
{
   myreduction0computemaxcourant_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp,s);
   hipDeviceSynchronize();
}
//reduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
//myreduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd,*d_wtemp, order, dir);

 //reductiona0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
  // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
    // fastcomputemaxc_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
hipDeviceSynchronize();

//(*p)->cmax=2.0;
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

//printf("cmax on device %.8f\n",(*p)->cmax);
//(*p)->cmax=0.0;
//hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
/*for(int i=0; i<dimp;i++)
{

if(((*wd)[i+(cfast*dimp)])>((*p)->cmax))
                    (*p)->cmax=(*wd)[i+(cfast*dimp)];
}
printf("cmax on cpu %.8f\n",(*p)->cmax);*/
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 /*for(int i=0; i<nit;i++)
{
 reduction0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
   // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
     hipDeviceSynchronize();
}*/



//    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

  // hipFree(*d_ttemp);
  //checkErrors("copy data from device");


 


}




int cucomputec(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computec_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputept(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

    computeptzero_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
     computept_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputepk(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computepk_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}


int cucomputepbg(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computepbg_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}







