#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/iotypes.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_i.cuh"
#include "../include/init_user_i.cuh"


//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));

   
 #ifdef USE_SAC_3D
   int kp;
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
		//b[i+j*(p->n[0])]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read

	    for(int f=0; f<NVAR; f++)
            { 		         
                          for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=0;
	    }



//	 __syncthreads();

			}

        	
	 __syncthreads();


    /* #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
     
               for(int f=vel1; f<NDERV; f++)
                    wd[fencode3_i(p,ii,f)]=0.0;
     

 __syncthreads(); */



     #ifdef USE_SAC_3D
      // if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[2]>1 && ii[0]<(p->n[0])-1 && ii[1]<(p->n[1])-1 && ii[2]<(p->n[2])-1)
         if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1])   && ii[2]<(p->n[2])) 
     #else
      // if((p->readini==0) && ii[0]>2 && ii[1]>2 && ii[0]<(p->n[0])-3 && ii[1]<(p->n[1])-3)  //this form for OZT test???? 
     
     
     //if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[0]<(p->n[0])-1 && ii[1]<(p->n[1])-1)  //this form for OZT test???? 
        if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1]))  //this form for BW test  //still issue here
     #endif
	{


            #ifdef ADIABHYDRO
		    if(i> (((p->n[0])/2)-2) && i<(((p->n[0])/2)+2) && j>(((p->n[1])/2)-2) && j<(((p->n[1])/2)+2) ) 
				w[fencode3_i(p,ii,rho)]=1.3;
            #else
                   // init_alftest (real *w, struct params *p,int i, int j)
                   // init_alftest(w,p,i,j);
                   // init_ozttest (real *w, struct params *p,int i, int j)
                   // init_ozttest(w,p,i,j);
                   // init_bwtest(w,p,i,j);

	           //default values for positions these may be updated by the initialisation routines
                   wd[fencode3_i(p,ii,delx1)]=(p->dx[0]);
		   wd[fencode3_i(p,ii,delx2)]=(p->dx[1]);
                   wd[fencode3_i(p,ii,pos1)]=(p->xmin[0])+ii[0]*(p->dx[0]);
		   wd[fencode3_i(p,ii,pos2)]=(p->xmin[1])+ii[1]*(p->dx[1]);
                 #ifdef USE_SAC_3D
		   wd[fencode3_i(p,ii,pos3)]=(p->xmin[2])+ii[2]*(p->dx[2]);
                   wd[fencode3_i(p,ii,delx3)]=(p->dx[2]);
                 #endif

                   init_user_i(w,p,ii);
           #endif

	

        }
	
	 __syncthreads();


       





     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
        /*for(int f=energyb; f<NVAR; f++)
             if(f != rhob)
                      w[fencode3_i(p,ii,f)]=0.0;*/
        //w[fencode3_i(p,ii,b2b)]=w[fencode3_i(p,ii,b3b)];
        for(int f=rho; f<NVAR; f++)
        {               
                  //wmod[fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];
                  //wmod[  (((3*(1+(p->rkon)))-1)*NVAR*dimp)+fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];              
                  dwn1[fencode3_i(p,ii,f)]=0;
                  for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                  {
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=w[fencode3_i(p,ii,f)];
                              //wmod[fencode3_i(p,ii,b2b)+ord*NVAR*dimp]=w[fencode3_i(p,ii,b3b)];
                  }
                            
        }

        for(int f=tmp1; f<NTEMP; f++)
                 wtemp[fencode3_i(p,ii,f)]=0;


}

 __syncthreads();



}



 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
//set up a temporary grid

__global__ void gridsetup_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp,kp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));
   kp=0;
   
 #ifdef USE_SAC_3D
 
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


     #ifdef USE_SAC_3D
       if(ii[0]>0 && ii[0]<(p->n[0]-1) && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
     {


        switch(dir)
        {

    case 0:
           wtemp2[encode3p2_i(p,ip+1,jp+1,kp+1,tmpnui)]=wd[fencode3_i(p,ii,pos1)];
    break;
    case 1:
           wtemp2[encode3p2_i(p,ip+1,jp+1,kp+1,tmpnui1)]=wd[fencode3_i(p,ii,pos2)];
    break;
    #ifdef USE_SAC_3D
           case 2:
                        wtemp2[encode3p2_i(p,ip+1,jp+1,kp+1,tmpnui2)]=wd[fencode3_i(p,ii,pos3)];
           break;
     #endif
           }
     }


        	
	 __syncthreads();




       





}




 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
//calculate the dx values

__global__ void setupdx_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp,kp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));

   
 #ifdef USE_SAC_3D
 
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

   //calculate the dx values


	    switch(dir)
	    {
		     case 0:
		     #ifdef USE_SAC_3D
		       if( ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 &&  ii[1]<(p->n[1])+1 && ii[2]>0 &&  ii[2]<(p->n[2])+1)
		     #else
		       if( ii[0]>0 && ii[0]<(p->n[0])+1  && ii[1]>0 && ii[1]<(p->n[1])+1)
		     #endif
	                wd[fencode3_i(p,ii,delx1)]=0.5*(wtemp2[encode3p2_i(p,ip+1,jp,kp,tmpnui)]-wtemp2[encode3p2_i(p,ip-1,jp,kp,tmpnui)]);
		     break;
	
		     case 1:
		     #ifdef USE_SAC_3D
		       if(ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 &&  ii[1]<(p->n[1])+1 && ii[2]>0 &&  ii[2]<(p->n[2])+1)
		     #else
		       if(ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 && ii[1]<(p->n[1])+1)
		     #endif
			wd[fencode3_i(p,ii,delx2)]=0.5*(wtemp2[encode3p2_i(p,ip,jp+1,kp,tmpnui)]-wtemp2[encode3p2_i(p,ip,jp-1,kp,tmpnui)]);
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:

		       if(ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 && ii[1]<(p->n[1])+1 && ii[2]>0 && ii[2]<(p->n[2])+1)
			wd[fencode3_i(p,ii,delx3)]=0.5*(wtemp2[encode3p2_i(p,ip,jp,kp+1,tmpnui)]-wtemp2[encode3p2_i(p,ip,jp,kp-1,tmpnui)]);
		     break;			
		     #endif
	     }
     
        	
	 __syncthreads();







       





}

 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
//intialise temporrary matrix needs t be completed
__global__ void zerotempv_parallel(struct params *p, real *w, real *wnew, real *wmod, 
real *dwn1,  real *wd, real *wtemp, real *wtemp1, real *wtemp2,  int dir)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  real maxt=0,max3=0, max1=0;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];


  
   int ip,jp;



  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
//int shift=order*NVAR*dimp;




//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {


        for(int f=d1; f<=d3; f++)
     #ifdef USE_SAC_3D
                 wtemp2[encode3p2_i(p,ii[0],ii[1],ii[2],tmpnui)]=0;
     #else
                 wtemp2[encode3p2_i(p,ii[0],ii[1],k,tmpnui)]=0;
     #endif

      if(i==((p->n[0])-1))
      {
        wtemp2[encode3p2_i(p,i+1,j,k,tmpnui)]=0;
        wtemp2[encode3p2_i(p,i+2,j,k,tmpnui)]=0;
      }
      if(j==((p->n[1])-1))
      {
          wtemp2[encode3p2_i(p,i,j+1,k,tmpnui)]=0;
          wtemp2[encode3p2_i(p,i,j+2,k,tmpnui)]=0;
      }

     #ifdef USE_SAC_3D
      if(k==((p->n[2])-1))
      {
          wtemp2[encode3p2_i(p,i,j,k+1,tmpnui)]=0;
          wtemp2[encode3p2_i(p,i,j,k+2,tmpnui)]=0;
      }

     #endif
      if(j==((p->n[1])-1)  && i==((p->n[0])-1))
      {
          for(int di=0; di<2; di++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_i(p,i+1+di,j+1+dj,k,tmpnui)]=0;
      }
     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1)  && k==((p->n[2])-1))
      {
          for(int di=0; di<2; di++)
             for(int dk=0; dk<2; dk++)
                   wtemp2[encode3p2_i(p,i+1+di,j,k+1+dk,tmpnui)]=0;
      }
      #endif

    

     #ifdef USE_SAC_3D
      if(j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_i(p,i,j+1+dj,k+1+dk,tmpnui)]=0;
      }
      #endif

     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1) && j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
               for(int di=0; di<2; di++)
                   wtemp2[encode3p2_i(p,i+1+di,j+1+dj,k+1+dk,tmpnui)]=0;
      }
      #endif

   }

}



__device__ __host__
int encodempiw (struct params *p,int ix, int iy, int iz, int field,int bound,int dim) {
  #ifdef USE_SAC_3D
    return (dim*(    4*NVAR*(         ((p->n[0])*(p->n[1]))+((p->n[1])*(p->n[2]))+((p->n[0])*(p->n[2]))   )           )+4*field*(         ((p->n[0])*(p->n[1]))+((p->n[1])*(p->n[2]))+((p->n[0])*(p->n[2]))   )+
bound*(         (dim==2)*((p->n[0])*(p->n[1]))   +  (dim==0)*((p->n[1])*(p->n[2]))  +   (dim==1)*((p->n[0])*(p->n[2]))    )+   (  (ix+iz*(p->n[0]))*(dim==1)+(iy+iz*(p->n[1]))*(dim==0)+(iz+ix*(p->n[2]))*(dim==2)    ));
  #else
    return (dim*(4*NVAR*((p->n[0])+(p->n[1])))+4*field*((p->n[0])+(p->n[1]))+bound*((dim==1)*(p->n[0])+(dim==0)*(p->n[1]))  +   (ix*(dim==1)+iy*(dim==0)));
  #endif
}

__device__ __host__
int encodempivisc (struct params *p,int ix, int iy, int iz, int bound,int dim) {
  #ifdef USE_SAC_3D
    return (dim*(    2*(         (((p->n[0])+2)*((p->n[1])+2))+(((p->n[1])+2)*((p->n[2])+2))+(((p->n[0])+2)*((p->n[2])+2))   )           )+
bound*(         (dim==2)*(((p->n[0])+2)*((p->n[1])+2))   +  (dim==0)*(((p->n[1])+2)*((p->n[2])+2))  +   (dim==1)*(((p->n[0])+2)*((p->n[2])+2))    )+   (  (ix+iz*((p->n[0])+2))*(dim==1)+(iy+iz*((p->n[1])+2))*(dim==0)+(iz+ix*((p->n[2])+2))*(dim==2)    ));
  #else
    return (   dim*(2*(  ((p->n[0])+2)+((p->n[1])+2)   ))      +bound*(    (dim==1)*((p->n[0])+2)+(dim==0)*((p->n[1])+2)  )  +   (ix*(dim==1)+iy*(dim==0))     );
  #endif
}



     __device__ __host__ void mpiwtogpu(struct params *p,real *d_w,real *d_wmod,real *d_mpiw,real *d_mpiwmod,int *ii, int var, int dim)
    {

             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=1+(p->n[0])-i;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=1+(p->n[1])-j;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=1+(p->n[0])-k;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }

     #endif
 


    }

__device__ __host__ void   mpivisctogpu(struct params *p,real *d_wtemp2,real *d_gmpivisc,int *ii,  int dim)
{
                                
               int i,j,k,bound,var;
              var=0;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 ) && dim==0)
                {              
                    bound=i;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                    
                }
                else if((( i==((p->n[0])+1)   ))  && dim==0)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }

              

                if((j==0) && dim==1)              
                {              
                    bound=j;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }            
                 else if((( j==((p->n[1])+1)   ))  && dim==1)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 ) && dim==2)              
                {              
                    bound=k;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }        
                 else if(((k==((p->n[2])+1)   ))  && dim==2)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }

     #endif
                               
                                
}

__device__ __host__ void   gputompivisc(struct params *p,real *d_wtemp2,real *d_gmpivisc,int *ii,  int dim)
{
                                
              int i,j,k,bound,var;
              var=0;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 ) && dim==0)
                {              
                    bound=i;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                    
                }
                else if((( i==((p->n[0])+1)   ))  && dim==0)               
                {
                    bound=1;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }

              

                if((j==0) && dim==1)              
                {              
                    bound=j;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }            
                 else if((( j==((p->n[1])+1)   ))  && dim==1)               
                {
                    bound=1;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 ) && dim==2)              
                {              
                    bound=k;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }        
                 else if(((k==((p->n[2])+1)   ))  && dim==2)               
                {
                    bound=1;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }

     #endif
                               
                                
}

     __device__ __host__ void gputompiw(struct params *p,real *d_w,real *d_wmod,real *d_mpiw,real *d_mpiwmod,int *ii, int var, int dim)
    {
             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];              
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=1+(p->n[0])-i;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];               
                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];              
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=1+(p->n[1])-j;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];               
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=1+(p->n[0])-k;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];               
                }

     #endif
 
 }

__global__ void  mpiwtogpu_parallel(struct params *p,real *d_w, real *d_wmod, real *d_mpiw, real *d_mpiwmod)
{

int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

#ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      for(int dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b3; f++)
     #else
     for(int dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b2; f++)
     #endif     
         #ifdef USE_SAC_3D
           if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
         #else
           if(i<((p->n[0])) && j<((p->n[1])))
         #endif           
                      mpiwtogpu(p,d_w,d_wmod,d_mpiw,d_mpiwmod,iia,f,dim);


 __syncthreads();

           
               
}


     __global__ void gputompiw_parallel(struct params *p,real *d_w,real *d_wmod,real *d_mpiw,real *d_mpiwmod,int order)
    {

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      for(dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b3; f++)
     #else
           for(dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b2; f++)
     #endif
             {
            
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{

 

                  gputompiw(p,d_w,d_wmod,d_mpiw,d_mpiwmod,iia,f,dim);

	}

               }

 __syncthreads();

}



     __global__ void gputompivisc_parallel(struct params *p,real *d_wtemp2,real *d_gmpivisc)
     {
               
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/((nj+2)*(ni+2));
   jp=(iindex-(kp*((nj+2)*(ni+2))))/(ni+2);
   ip=iindex-(kp*(nj+2)*(ni+2))-(jp*(ni+2));
#else
    jp=iindex/(ni+2);
   ip=iindex-(jp*(ni+2));
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];

     #else

     #endif
           for(dim=0; dim<NDIM;dim++)
             {
            
         #ifdef USE_SAC_3D
      if(i<(((p->n[0])+2)) && j<(((p->n[1])+2))  && k<(((p->n[2])+2)))
     #else
       if(i<(((p->n[0])+2)) && j<(((p->n[1])+2)))
     #endif           
	{

 

                  gputompivisc(p,d_wtemp2,d_gmpivisc,iia,dim);

	}

               }

 __syncthreads();
              
               }    
     
     
    __global__ void  mpivisctogpu_parallel(struct params *p,real *d_wtemp2,real *d_gmpivisc)
    {
               
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/((nj+2)*(ni+2));
   jp=(iindex-(kp*((nj+2)*(ni+2))))/(ni+2);
   ip=iindex-(kp*(nj+2)*(ni+2))-(jp*(ni+2));
#else
    jp=iindex/(ni+2);
   ip=iindex-(jp*(ni+2));
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];

     #else

     #endif
           for(dim=0; dim<NDIM;dim++)
             {
            
         #ifdef USE_SAC_3D
      if(i<(((p->n[0])+2)) && j<(((p->n[1])+2))  && k<(((p->n[2])+2)))
     #else
       if(i<(((p->n[0])+2)) && j<(((p->n[1])+2)))
     #endif           
	{

 

                  mpivisctogpu(p,d_wtemp2,d_gmpivisc,iia,dim);

	}

               }

 __syncthreads();
               
               
}



/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, struct bparams **bp,real **w, real **wnew, real **wd, struct state **state, struct params **d_p, struct bparams **d_bp,real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  int dir;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct bparams *adbp;
  struct state *ads;
 
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

if(((*p)->rkon)==1)
  hipMalloc((void**)d_wmod, 6*NVAR*dimp*sizeof(real));
else
  hipMalloc((void**)d_wmod, 3*NVAR*dimp*sizeof(real));

  hipMalloc((void**)d_dwn1, NVAR*dimp*sizeof(real));
  hipMalloc((void**)d_wd, NDERV*dimp*sizeof(real));
  hipMalloc((void**)d_wtemp, NTEMP*dimp*sizeof(real));


  #ifdef USE_SAC
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
  #endif
  #ifdef USE_SAC_3D
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)* (((*p)->n[2])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real));
  #endif

  hipMalloc((void**)&adw, NVAR*dimp*sizeof(real));
  hipMalloc((void**)&adwnew, NVAR*dimp*sizeof(real));

  hipMalloc((void**)&adbp, sizeof(struct bparams));
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->n[1]);

   // *d_b=adb;
    *d_bp=adbp;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;

     
printf("allocating %d %d %d %d\n",dimp,(*p)->n[0],(*p)->n[1],(*p)->n[2]);
    hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(*d_wd, *wd, NDERV*dimp*sizeof(real), hipMemcpyHostToDevice);

   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
    printf("here\n");
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2);
     hipDeviceSynchronize();
     
     //copy data back to cpu so we can compute and update the grid (on the cpu)
    hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
    //setup the grid and dx values here


    hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);


 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
 /*for(dir=0; dir<NDIM; dir++)
 {
     zerotempv_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2,dir);
     hipDeviceSynchronize();     
     gridsetup_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2,dir);
     hipDeviceSynchronize();
     setupdx_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2,dir);
     hipDeviceSynchronize();
  }*/

	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

        // printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->n[0])+((*p)->n[0])*((*p)->n[1])*b1]));



  return 0;



}

/*! Cartesian or polar grid. Determine x at the boundaries.
! Determine often needed combinations of x, such as dx or dvolume.
! Determine variables for axial symmetry
!
! ixe          - edge coordinate of the grid touching the boundary region
! ixf          - coordinate inside of ixe
! qx           - x with an extended index range for calculation of dx   */

int initgrid(struct params **p, real **w, real **wnew,   struct state **state, real **wd, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{
    real *ttemp2;
    int ii[NDIM];
    int ii1[3],ii2[3],ix;
    int ip,jp,kp,kpo;
    int dir,dir1,dir2;
    int ixmin,ixmax,ixe,ixf;
    real *wda=*wd;
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
 #ifdef USE_SAC_3D
 
   dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
    kp=0;
    printf("called initgrid\n");
    

    for(int i=0;i<3;i++)
    {
       ii1[i]=0;
       ii2[i]=0;
    }
    #ifdef USE_SAC
    ttemp2=(real *) malloc( (NTEMP2+2)*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
    #endif
    #ifdef USE_SAC_3D
    ttemp2=(real *)malloc((NTEMP2+2)*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real));
    #endif
    
   	hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
     for(dir=0;dir<NDIM;dir++)
     for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
     		     #ifdef USE_SAC_3D
                   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
                 #endif
                 {
                        ip=ii[0];
                        jp=ii[1];
         		     #ifdef USE_SAC_3D
                       kp=ii[2];
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	                ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui)]= 0;
		     break;
	
		     case 1:
			 ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui1)]= 0;
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			 ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui2)]= 0;
		     break;			
		     #endif
	     }
      }	
 

     kp=1;
     kpo=0;
     for(dir=0;dir<NDIM;dir++)
        for(ii[0]=1; ii[0]<((*p)->n[0])+1; ii[0]++)
           for(ii[1]=1; ii[1]<((*p)->n[1])+1; ii[1]++)
		#ifdef USE_SAC_3D
		   for(ii[2]=1; ii[2]<((*p)->n[2])+1; ii[2]++)
		#endif
                {
                        ip=ii[0];
                        jp=ii[1];
         		     #ifdef USE_SAC_3D
                       kp=ii[2];
                       kpo=kp;
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	                ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui)]= (wda[encode3_i(*p,ip-1,jp-1,kp-1,pos1)]);
		     break;
	
		     case 1:
			 ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui1)]= (wda[(encode3_i(*p,ip-1,jp-1,kp-1,pos2))]);
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			 ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui2)]= (wda[(encode3_i(*p,ip-1,jp-1,kp-1,pos3))]);
		     break;			
		     #endif
	     }
      }	


  	
   	//update grid edges
     kp=0;
     for(dir=0;dir<NDIM;dir++)
     {
                
                       
	    switch(dir)
	    {
		     case 0:
                       ixmax=((*p)->n[0])+1;//ixGmax1+1; 
                       ixmin=((*p)->n[0])+1;//ixmin1=ixGmax1+1                      
                       ixe=ixmin-1; 
                       ixf=ixe-1;


                       //upper layers
			     for(dir1=0;dir1<NDIM;dir1++)
			     {
				     for(ii[0]=ixmin; ii[0]<=ixmax; ii[0]++)
				     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
				                        ix=ii[0];
                                                        ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[0]=ixe;
                                                       ii2[0]=ixf; 
                                                       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						      //ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (wda[fencode3_i(*p,ii1,pos1+dir1)]);
						  }

				}



                      //lower layers

                       ixmin=0;//ixmin1=ixGmin1-1;
                       ixmax=0;//ixmax1=ixGmin1-1                   
                       ixe=ixmax+1; 
                       ixf=ixe+1;

			     for(dir1=0;dir1<NDIM;dir1++)
			     {
				     for(ii[0]=ixmin; ii[0]<=ixmax; ii[0]++)
				     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
							ix=ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[0]=ixe;
                                                       ii2[0]=ixf; 
    ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
// ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]= (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])+ (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
   // qx(ix,ixmin2:ixmax2,jdim)=(1+abs(ixe-ix))*qx(ixe,ixmin2:ixmax2,jdim)- abs(ixe-ix) *qx(ixf,ixmin2:ixmax2,jdim)

						  }

				}
		     break;
	
		     case 1:


                       ixmax=((*p)->n[1])+1;//ixGmax1+1; 
                       ixmin=((*p)->n[1])+1;//ixGmax1+1;                      
                       ixe=ixmin-1; 
                       ixf=ixe-1;


                       //upper layers
			     for(dir1=0;dir1<NDIM;dir1++)
			     {
                 for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
				     for(ii[1]=ixmin; ii[1]<=ixmax; ii[1]++)
				     
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							ix=jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[1]=ixe;
                                                       ii2[1]=ixf; 
						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						      //ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (wda[fencode3_i(*p,ii1,pos1+dir1)]);
						  }

				}



                      //lower layers

                       ixmin=0;//ixmin1=ixGmin1-1;
                       ixmax=0;//ixmax1=ixGmin1-1                    
                       ixe=ixmax+1; 
                       ixf=ixe+1;

			     for(dir1=0;dir1<NDIM;dir1++)
			     {
			         for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)	
				     for(ii[1]=ixmin; ii[1]<=ixmax; ii[1]++)
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							ix=jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[1]=ixe;
                                                       ii2[1]=ixf; 
						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						  }

				}




		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:


                       ixmax=((*p)->n[2])+1;//ixGmax1+1; 
                       ixmin=((*p)->n[2])+1;//ixGmax1+1;                      
                       ixe=ixmin-1; 
                       ixf=ixe-1;


                       //upper layers
			     for(dir1=0;dir1<NDIM;dir1++)
			     {
                 for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
                 for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
				     
				     		 #ifdef USE_SAC_3D
						  
			        for(ii[2]=ixmin; ii[2]<=ixmax; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       ix=kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[2]=ixe;
                                                       ii2[2]=ixf; 
						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						      //ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (wda[fencode3_i(*p,ii1,pos1+dir1)]);
						  }

				}



                      //lower layers

                       ixmin=0;//ixmin1=ixGmin1-1;
                       ixmax=0;//ixmax1=ixGmin1-1                    
                       ixe=ixmax+1; 
                       ixf=ixe+1;

			     for(dir1=0;dir1<NDIM;dir1++)
			     {
			         for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
                     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)	
				     
				     		 #ifdef USE_SAC_3D
						   
						    for(ii[2]=ixmin; ii[2]<=ixmax; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       ix=kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[2]=ixe;
                                                       ii2[2]=ixf; 
						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						  }

				}



		     break;			
		     #endif
	     }
      }	






   	//calculate dx
  kp=0;
  kpo=0;

                   for(dir=0;dir<NDIM;dir++)
                 {

  for(ii[0]=1; ii[0]<((*p)->n[0])+1; ii[0]++)
     for(ii[1]=1; ii[1]<((*p)->n[1])+1; ii[1]++)
  //for(ii[0]=0; ii[0]<((*p)->n[0]); ii[0]++)
  //   for(ii[1]=0; ii[1]<((*p)->n[1]); ii[1]++)

     		     #ifdef USE_SAC_3D
                   for(ii[2]=1; ii[2]<((*p)->n[2])+1; ii[2]++)
                 #endif
{

                        ip=ii[0];
                        jp=ii[1];
         		     #ifdef USE_SAC_3D
                       
                       kp=ii[2];
                        kpo=kp-1;
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	               // (wda[(encode3_i(*p,ip-1,jp-1,kpo,delx1))])=/*(*p)->dx[0];//*/0.5*(ttemp2[encode3p2_i(*p,ip+1,jp,kp,tmpnui)]-ttemp2[encode3p2_i(*p,ip-1,jp,kp,tmpnui)]);
                  (wda[(encode3_i(*p,ip-1,jp-1,kpo,delx1))])=/*(*p)->dx[0];//*/0.5*(ttemp2[encode3p2_i(*p,ip+1,jp,kp,tmpnui)]-ttemp2[encode3p2_i(*p,ip-1,jp,kp,tmpnui)]);
	              //  if(ip==128  && jp==128 && kp==128)
                      //  printf("delx 0 %d %d %d %16.20f  %16.20f   %16.20f \n",ii[0]-1,ii[1]-1,ii[2]-1,wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx2))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx3))]);
		     break;
	
		     case 1:
			(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx2))])=/*(*p)->dx[1];//*/0.5*(ttemp2[encode3p2_i(*p,ip,jp+1,kp,tmpnui1)]-ttemp2[encode3p2_i(*p,ip,jp-1,kp,tmpnui1)]);
	               // if(ip==128  && jp==128 && kp==128)
                       //   printf("delx 1 %d %d %d %16.20f  %16.20f   %16.20f \n",ii[0]-1,ii[1]-1,ii[2]-1,wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx2))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx3))]);

		        //printf("delx2 %d %d %g ",ii[0],ii[1],wda[(fencode3_i(*p,ii,delx2))]);
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx3))])=0.5*(ttemp2[encode3p2_i(*p,ip,jp,kp+1,tmpnui2)]-ttemp2[encode3p2_i(*p,ip,jp,kp-1,tmpnui2)]);
	              //  if(ip==128  && jp==128 && kp==128)
                      //  printf("delx 2 %d %d %d %16.20f  %16.20f   %16.20f \n",ii[0]-1,ii[1]-1,ii[2]-1,wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx2))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx3))]);

		     break;			
		     #endif
	     }
      }
  printf("\n");
}


printf("dx=%g dy=%g\n",(*p)->dx[0], (*p)->dx[1] );


kp=0;

     for(dir=0;dir<NDIM;dir++)
        for(ii[0]=0; ii[0]<((*p)->n[0]); ii[0]++)
           for(ii[1]=0; ii[1]<((*p)->n[1]); ii[1]++)
		#ifdef USE_SAC_3D
		   for(ii[2]=0; ii[2]<((*p)->n[2]); ii[2]++)
		#endif
                {
                        ip=ii[0]+1;
                        jp=ii[1]+1;
         		     #ifdef USE_SAC_3D
                       kp=ii[2]+1;
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	                 (wda[fencode3_i(*p,ii,pos1)])=ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui)];
                      //  if(ip==1)
                      //  printf("delx 0 %d %d %16.20f  %16.20f \n",ii[0],ii[1],wda[(encode3_i(*p,ip-1,jp-1,kp,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp,delx2))]);
		     break;
	
		     case 1:
			  (wda[(fencode3_i(*p,ii,pos2))])=ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui1)];
//if(ip==1)
                 //       printf("delx 1 %d %d %16.20f  %16.20f \n",ii[0],ii[1],wda[(encode3_i(*p,ip-1,jp-1,kp,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp,delx2))]);

		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			  (wda[(fencode3_i(*p,ii,pos3))])=ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui2)];
		     break;			
		     #endif
	     }
      }	

     kp=0;
     //for(dir=0;dir<NDIM;dir++)
       /* for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
           for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
             {

                        ip=ii[0];
                        jp=ii[1];
                if(ii[0]==0)
                printf("delx 0 %d %d %16.20f  %16.20f \n",ii[0],ii[1],ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui)],ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui1)]);

              }*/

    hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(*d_wd, *wd, NDERV*dimp*sizeof(real), hipMemcpyHostToDevice);
  

    free(ttemp2);
  return 0;



}


#ifdef USE_MPI

//prepare data buffers used to copy data between gpu and cpu
//this will update only the ghost cells transferred between the CPU's


int cuinitmpibuffers(struct params **p,real **w, real **wmod, real **temp2, real **gmpivisc,   real **gmpiw, real **gmpiwmod, struct params **d_p,   real **d_w, real **d_wmod,real **d_wtemp2,    real **d_gmpivisc,   real **d_gmpiw, real **d_gmpiwmod)
{

  int szw,  szvisc;
  #ifdef USE_SAC
  real *dt;
  
  szw=4*(  ((*p)->n[1])  +  ((*p)->n[0])   );
  szvisc=4*(  (((*p)->n[1])+2 )  +  (((*p)->n[0]) +2 )  );
 dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2),sizeof(real));

  #endif
  #ifdef USE_SAC_3D
  
  szw=4*NVAR*(  ((*p)->n[1])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[1])  );
  szvisc=4*NVAR*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)  +  (((*p)->n[0])+2)*(((*p)->n[2])+2)  +  (((*p)->n[0])+2)*(((*p)->n[1])+2)  );    
  dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2),sizeof(real));
  #endif






  temp2=&dt;
  gmpiwmod=(real **)malloc(szw*sizeof(real));
  gmpiw=(real **)malloc(szw*sizeof(real));
  gmpivisc=(real **)malloc(szvisc*sizeof(real));
  
  
  hipMalloc((void**)d_gmpiwmod, NVAR*szw*sizeof(real));
  hipMalloc((void**)d_gmpiw, NVAR*szw*sizeof(real));
  hipMalloc((void**)d_gmpivisc, szvisc*sizeof(real));
  return 0;
}

//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopywtompiw(struct params **p,real **w, real **wmod,    real **gmpiw, real **gmpiwmod, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw, real **d_gmpiwmod, int order)
{
     int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;

     int szbuf;
     int dimp=(((*p)->n[0]))*(((*p)->n[1]));
     
     
   
     #ifdef USE_SAC_3D  
       dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
     #endif 
     int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif

    // for(var=0; var<NVAR; var++)
    //   for(dim=0;dim<NDIM;dim++)
     gputompiw_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw,*d_gmpiwmod,order);
     hipDeviceSynchronize();
     hipMemcpy(*gmpiwmod, *d_gmpiwmod, NVAR*szbuf*sizeof(real), hipMemcpyDeviceToHost);
     hipMemcpy(*gmpiw, *d_gmpiw, NVAR*szbuf*sizeof(real), hipMemcpyDeviceToHost);
     
     
//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;

                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                                                                     
                      // *(wmod+encode3_i(*p,ii,var))=*(gmpiwmod+encodempiw(*p,i1,i2,i3,var,bound,dim));              
                      // (*w)[encode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
         i3=bound*(bound<2)+( ((*p)->n[2])-(bound-1) )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    

}

//copy mpi recv buffer to gpu memory     
int cucopywfrommpiw(struct params **p,real **w, real **wmod,    real **gmpiw, real **gmpiwmod, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw, real **d_gmpiwmod, int order)
{
       int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;     
       int szbuf;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

      //copy data from w and wmod to correct gmpiw and gmpiwmod

//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];

                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;  

                       (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];

                  }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      

                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
         i3=bound*(bound<2)+(((*p)->n[2])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                    }                            
                       break;                       
            #endif             
             }
                                     
         }    //encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;     

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
  
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                   }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;


                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
         i3=bound*(bound<2)+(((*p)->n[2])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 


                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                   }                            
                       break;                       
            #endif             
             }
                                     
         }    




   	 hipMemcpy(*d_gmpiw, *gmpiw, NVAR*szbuf*sizeof(real), hipMemcpyHostToDevice);     
   	 hipMemcpy(*d_gmpiwmod, *gmpiwmod, NVAR*szbuf*sizeof(real), hipMemcpyHostToDevice);     

     mpiwtogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw,*d_gmpiwmod);
     hipDeviceSynchronize();
}

//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopytompivisc(struct params **p,real **temp2, real **gmpivisc,  struct params **d_p,real **d_wtemp2,    real **d_gmpivisc)
{


     int szbuf;
     int dim,bound,var=0;
     int i1,i2,i3;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
             int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
     gputompivisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wtemp2,*d_gmpivisc);
     hipDeviceSynchronize();
     hipMemcpy(*gmpivisc, *d_gmpivisc, NVAR*szbuf*sizeof(real), hipMemcpyDeviceToHost);

     //copy data to correct area in temp2
//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<2;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(((*p)->n[0])+1);
         for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {     
                        
          //i1=(p->n[0])+1;
         
          //temp2[encode3p2_sacmpi (p,i1, i2, i3, tmpnui)]=gmpitgtbufferr[0][i2+i3*((p->n[1])+2)];
          //temp2[encode3p2_sacmpi (p,0, i2, i3, tmpnui)]=gmpitgtbufferl[0][i2+i3*((p->n[1])+2)];
         
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }
            #else
         i3=0;
         i1=bound*(((*p)->n[0])+1);
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(((*p)->n[1])+1);
         for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }

            #else
         i3=0;
         i2=bound*(((*p)->n[1])+1);
                  for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  {
                                                                     
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
                  i3=bound*(((*p)->n[2])+1);
        for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                                                              
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    

}

//copy mpi recv buffer to gpu memory     
int cucopyfrommpivisc(struct params **p,real **temp2,real **gmpivisc,  struct params **d_p,real **d_wtemp2,    real **d_gmpivisc)
{
      int dim,bound,var=0;
     int i1,i2,i3;      
       int szbuf;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif

      //copy data from temp2 to gmpivisc
             for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<2;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(((*p)->n[0])+1);
         for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {     
                        
          //i1=(p->n[0])+1;
         
          //temp2[encode3p2_sacmpi (p,i1, i2, i3, tmpnui)]=gmpitgtbufferr[0][i2+i3*((p->n[1])+2)];
          //temp2[encode3p2_sacmpi (p,0, i2, i3, tmpnui)]=gmpitgtbufferl[0][i2+i3*((p->n[1])+2)];
         
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }
            #else
         i3=0;
         i1=bound*(((*p)->n[0])+1);
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(((*p)->n[1])+1);
         for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }

            #else
         i3=0;
         i2=bound*(((*p)->n[1])+1);
                  for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  {
                                                                     
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
                  i3=bound*(((*p)->n[2])+1);
        for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                                                              
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    


   	 hipMemcpy(*d_gmpivisc, *gmpivisc, NVAR*szbuf*sizeof(real), hipMemcpyHostToDevice);     

     mpivisctogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wtemp2,*d_gmpivisc);
     hipDeviceSynchronize();
}


#endif



