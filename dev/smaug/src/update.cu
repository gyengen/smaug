#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_u.cuh"


__device__ __host__
int updatestate (struct params *p, struct state *s, real *w ,int *ii, int field) {

  int status=0;
                      // atomicExch(&(p->cmax),(wd[fencode3_pre(p,ii,soundspeed)]));
                    switch(field)
                    {
                      case rho:
                    	s->rho=s->rho+(w[fencode3_u(p,ii,field)]);
		      break;
                      case mom1:
                    	s->m1=s->m1+(w[fencode3_u(p,ii,field)]);
		      break;
                      case mom2:
                    	s->m2=s->m2+(w[fencode3_u(p,ii,field)]);
		      break;
                      /*case mom3:
                    	s->m3=s->m3+(w[fencode3_u(p,ii,field)]);
		      break;*/
                      case energy:
                    	s->e=s->e+(w[fencode3_u(p,ii,field)]);
		      break;
                      case b1:
                    	s->b1=s->b1+(w[fencode3_u(p,ii,field)]);
		      break;
                      case b2:
                    	s->b2=s->b2+(w[fencode3_u(p,ii,field)]);
		      break;
                      /*case b3:
                    	s->b3=s->b3+(w[fencode3_u(p,ii,field)]);
		      break;*/
                    };
  return status;
}



__global__ void update_parallel(struct params *p, struct state *s, real *w, real *wmod)
{

   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,f;
  int index,k;
  __shared__ int ntot;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  //real g=p->g;
  real *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     



//int shift=order*NVAR*dimp;

  h=w+dimp*rho;
  u=w+dimp*mom1;
  v=w+dimp*mom2;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
           for( f=rho; f<=b3; f++)
     #else
           for( f=rho; f<=b2; f++)
     #endif
             {  
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{
            
                  w[fencode3_u(p,iia,f)]=wmod[fencode3_u(p,iia,f)];

	}


}

__syncthreads(); 







  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_u(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}
int cuupdate(struct params **p, real **w, real **wmod,real **wtemp2, struct state **state,struct params **d_p, real **d_w, real **d_wmod, real ** d_wtemp2, struct state **d_state, int step)
//int cuupdate(struct params **p, real **w, real **wmod, real **wd, real **temp2, struct state **state,
//             struct params **d_p, real **d_w, real **d_wmod, real **d_wtemp2, struct state **d_state, int step)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
    dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
  // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
     update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_state,*d_w,*d_wmod);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
//following comments removed from if def pragmas  if
//using MPI and copying all cell data to host (how slow!?)
//#ifdef USE_MPI

//#else
    if((step%((*p)->cfgsavefrequency))==0)
//#endif
    {

//following commentes removed from section if
//using MPI and copying all cell data to host (how slow!?)
/*#ifdef USE_MPI
    hipMemcpy(*wmod, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
    #ifdef USE_SAC_3D  
           hipMemcpy(*wtemp2, *d_wtemp2,NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real), hipMemcpyDeviceToHost);
    #else
       hipMemcpy(*wtemp2, *d_wtemp2,NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real), hipMemcpyDeviceToHost);
    #endif

#endif */ 
    hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

    //hipMemcpy(*wnew, *d_wd, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);

   hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
    }

//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


int cufinish(struct params **p, real **w, real **wnew, struct state **state, struct params **d_p,struct bparams **d_bp, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{
  

 //hipMemcpy(*w, *d_w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  checkErrors_u("copy data from device");


  hipFree(*d_p);
  hipFree(*d_bp);
//  hipFree(*d_state);

  hipFree(*d_w);
  hipFree(*d_wnew);
 // hipFree(*d_u);

  hipFree(*d_wmod);
  hipFree(*d_dwn1);
  hipFree(*d_wd);
  hipFree(*d_wtemp);
  hipFree(*d_wtemp1);
  hipFree(*d_wtemp2);
  




}

  #ifdef USE_MPI

int cufinishmpi(struct params **p,real **w, real **wmod, real **temp2, real **gmpivisc,   real **gmpiw, real **gmpiwmod, struct params **d_p,   real **d_w, real **d_wmod,real **d_wtemp2,    real **d_gmpivisc,   real **d_gmpiw, real **d_gmpiwmod)
{
  

 //hipMemcpy(*w, *d_w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  checkErrors_u("copy data from device");


  hipFree(*d_gmpiw);
  hipFree(*d_gmpiwmod);
  hipFree(*d_gmpivisc);

  free(*gmpiw);
  free(*gmpiwmod);
  free(*gmpivisc);
  free(*temp2);
}
#endif
